#include "hip/hip_runtime.h"

#include "dspk.h"

using namespace std;

namespace kso {

namespace img {

namespace dspk {

np::ndarray locate_noise_3D(const np::ndarray & cube, float std_dev, uint k_sz, uint Niter){


	kso::util::enum_device();

	// shape of input data
	uint sz_t = cube.get_shape()[0];
	uint sz_y = cube.get_shape()[1];
	uint sz_l = cube.get_shape()[2];
	uint sz = sz_t * sz_y * sz_l;

	// GPU information
	uint device = 0;
	float mem_fill = 0.5;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	size_t tot_mem = deviceProp.totalGlobalMem;
	size_t mem = tot_mem * mem_fill;

	// calculate chunking of input data
	uint n_threads = 1;		// Number of host threads
	uint n_buf = 6;		// number of unique buffers. THIS NUMBER IS HARDCODED. MAKE SURE TO CHANGE IF NEEDED!
	uint t_mem = mem / n_threads;	// Amount of memory per thead
	uint c_mem = t_mem / n_buf;		// Amount of memory per chunk per thread
	uint f_mem = sz_y * sz_l * sizeof(float); 	// Amount of memory occupied by a single frame (spectra / space)
	uint csz_t = c_mem / f_mem;		// Max number of frames per chunk
	uint N_t = ceil((float) (sz_t) / (float) (csz_t));	// Number of chunks per observation


	// extract float data from numpy array
	float * dt = (float *) cube.get_data();

	// initialize goodmap
	float * gm = new float[sz];
	float * gdev = new float[sz];
	float * nsd = new float[sz];
	fill(gm, gm + sz, 1.0);



	// storage for the number of bad pixels found on each iteration
	uint newBad = 0;
	uint totBad = 0;

	// allocate pointers for device data
	float * dt_d, * gm_d, * gdev_d, *nsd_d, *tmp_d, *norm_d;
	uint * newBad_d;

	// allocate memory on device
	CHECK(hipMalloc((float **) &dt_d, sz * sizeof(float)));
	CHECK(hipMalloc((float **) &gm_d, sz * sizeof(float)));
	CHECK(hipMalloc((float **) &gdev_d, sz * sizeof(float)));
	CHECK(hipMalloc((float **) &nsd_d, sz * sizeof(float)));
	CHECK(hipMalloc((float **) &tmp_d, sz * sizeof(float)));
	CHECK(hipMalloc((float **) &norm_d, sz * sizeof(float)));
	CHECK(hipMalloc((uint **) &newBad_d, sizeof(uint)));

	// copy memory to device
	CHECK(hipMemcpy(dt_d, dt, sz * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(gm_d, gm, sz * sizeof(float), hipMemcpyHostToDevice));;

	// number of blocks and threads
	dim3 threads(sz_l, 1, 1);
	dim3 blocks(1, sz_y, sz_t);


	// Number of identification iterations
	for(uint iter = 0; iter < Niter; iter++){

		newBad = 0;	// reset the number of bad pixels found for this iteration
		CHECK(hipMemcpy(newBad_d, &newBad, sizeof(uint), hipMemcpyHostToDevice));

		kso::img::dspk::calc_norm_0<<<blocks, threads>>>(norm_d, gm_d, sz3, k_sz);
		kso::img::dspk::calc_norm_1<<<blocks, threads>>>(tmp_d, norm_d, sz3, k_sz);
		kso::img::dspk::calc_norm_2<<<blocks, threads>>>(norm_d, tmp_d, sz3, k_sz);

		kso::img::dspk::calc_gdev_0<<<blocks, threads>>>(gdev_d, dt_d, gm_d, sz3, k_sz);
		kso::img::dspk::calc_gdev_1<<<blocks, threads>>>(tmp_d, gdev_d, sz3, k_sz);
		kso::img::dspk::calc_gdev_2<<<blocks, threads>>>(gdev_d, tmp_d, dt_d, gm_d, norm_d, sz3, k_sz);

		kso::img::dspk::calc_nsd_0<<<blocks, threads>>>(nsd_d, gdev_d, sz3, k_sz);
		kso::img::dspk::calc_nsd_1<<<blocks, threads>>>(tmp_d, nsd_d, sz3, k_sz);
		kso::img::dspk::calc_nsd_2<<<blocks, threads>>>(nsd_d, tmp_d, norm_d, sz3, k_sz);

		kso::img::dspk::calc_gm<<<blocks, threads>>>(gm_d, gdev_d, nsd_d, std_dev, newBad_d, sz3, k_sz);



		CHECK(hipDeviceSynchronize());


		CHECK(hipMemcpy(&newBad, newBad_d, sizeof(uint), hipMemcpyDeviceToHost));
		cout << "Iteration " << iter << ": found " << newBad << " bad pixels\n";
		totBad = totBad + newBad;

	}

	// copy back from devicecudaMemcpyDeviceToHost
	CHECK(hipMemcpy(gm, gm_d, sz * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(gdev, gdev_d, sz * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(nsd, nsd_d, sz * sizeof(float), hipMemcpyDeviceToHost));

	cout << "Total bad pixels: " << totBad << endl;

	// stride of input data
	uint n_t = cube.get_strides()[0];
	uint n_y = cube.get_strides()[1];
	uint n_l = cube.get_strides()[2];

	// prepare to return Numpy array
	p::object gm_own = p::object();
	p::tuple gm_stride = p::make_tuple(n_t, n_y, n_l);
	p::tuple gm_shape = p::make_tuple(sz_t, sz_y, sz_l);
	np::dtype gm_type = np::dtype::get_builtin<float>();
	np::ndarray gm_arr = np::from_data(gm, gm_type, gm_shape, gm_stride, gm_own);


	return gm_arr;


}









}

}

}





BOOST_PYTHON_MODULE(libkso_cuda){

	//	Py_Initialize();
	np::initialize();   // only needed if you use numpy in the interface

	boost::python::def("locate_noise_3D", kso::img::dspk::locate_noise_3D);

}


