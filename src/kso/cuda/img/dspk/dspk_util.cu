#include "hip/hip_runtime.h"

#include "dspk_util.h"

namespace kso {

namespace img {

namespace dspk {


buf::buf(float * data, float * goodmap, dim3 data_sz, uint kern_sz, uint n_threads){

	float mem_fill = 0.5;

	dt = data;
	gm = goodmap;

	sz = data_sz;
	sz3 = sz.x * sz.y * sz.z;

	st.x = 1;
	st.y = sz.x * st.x;
	st.z = sz.y * st.y;

	ksz = kern_sz;

	// Determine how much memory is available on this device
	uint device = 0;
	uint mem = mem_fill * kso::util::get_device_mem(device);

	// calculate chunking of input data
	//	uint n_threads = 2;		// Number of host threads
	uint n_buf = 6;		// number of unique buffers. THIS NUMBER IS HARDCODED. MAKE SURE TO CHANGE IF NEEDED!
	uint t_mem = mem / n_threads;	// Amount of memory per thead
	uint c_mem = t_mem / n_buf;		// Amount of memory per chunk per thread
	uint f_mem = sz.y * sz.x * sizeof(float); 	// Amount of memory occupied by a single frame (spectra / space)

	// save chunk sizes
	csz.x = sz.x;
	csz.y = sz.y;
	csz.z = c_mem / f_mem;		// Max number of frames per chunk
	csz3 = csz.z * csz.x * csz.y;		// number of elements in chunk
	cst.x = 1;
	cst.y = csz.x * cst.x;
	cst.z = csz.y * cst.y;

	printf("Total memory allocated: %.0f MiB\n", mem / pow(2,20) );
	printf("Memory per thread: %.0f MiB\n", t_mem / pow(2,20));
	printf("Memory per chunk: %.0f MiB\n", c_mem / pow(2,20));
	printf("Memory per frame: %.3f MiB\n", f_mem / pow(2,20));
	printf("Number of frames per chunk: %d\n", csz.z);

	// allocate host page-locked memory
	CHECK(hipHostRegister(dt, sz3 * sizeof(float), hipHostRegisterDefault));
	CHECK(hipHostRegister(gm, sz3 * sizeof(float), hipHostRegisterDefault));
	CHECK(hipHostAlloc(&newBad, sizeof(uint), hipHostRegisterDefault));

	// allocate memory on device
	CHECK(hipMalloc((float **) &dt_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &gm_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &gdev_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &nsd_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &tmp_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &norm_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((uint **) &newBad_d, sizeof(uint)));

	// calculate offset for kernel
	ks2 = ksz / 2;

	// calculate striding
	S = new kso::util::stride(sz.z, csz.z, 2 * ks2, cst.z);



}

buf::buf(std::string path, uint max_sz, uint kern_sz, uint n_threads){

	// allocate host page-locked memory
	CHECK(hipHostAlloc(&dt, max_sz * sizeof(float), hipHostRegisterDefault));
	CHECK(hipHostAlloc(&gm, sz3 * sizeof(float), hipHostRegisterDefault));
	CHECK(hipHostAlloc(&newBad, sizeof(uint), hipHostRegisterDefault));

	sz = instrument::IRIS::read_fits_raster(path, dt);

	sz3 = sz.x * sz.y * sz.z;

	st.x = 1;
	st.y = sz.x * st.x;
	st.z = sz.y * st.y;

	sb.x = st.x * sizeof(float);
	sb.y = st.y * sizeof(float);
	sb.z = st.z * sizeof(float);

	ksz = kern_sz;

	// Determine how much memory is available on this device
	float mem_fill = 0.5;
	uint device = 0;
	uint mem = mem_fill * kso::util::get_device_mem(device);

	// calculate chunking of input data
	//	uint n_threads = 2;		// Number of host threads
	uint n_buf = 6;		// number of unique buffers. THIS NUMBER IS HARDCODED. MAKE SURE TO CHANGE IF NEEDED!
	uint t_mem = mem / n_threads;	// Amount of memory per thead
	uint c_mem = t_mem / n_buf;		// Amount of memory per chunk per thread
	uint f_mem = sz.y * sz.x * sizeof(float); 	// Amount of memory occupied by a single frame (spectra / space)

	// save chunk sizes
	csz.x = sz.x;
	csz.y = sz.y;
	csz.z = c_mem / f_mem;		// Max number of frames per chunk
	csz3 = csz.z * csz.x * csz.y;		// number of elements in chunk
	cst.x = 1;
	cst.y = csz.x * cst.x;
	cst.z = csz.y * cst.y;

	printf("Total memory allocated: %.0f MiB\n", mem / pow(2,20) );
	printf("Memory per thread: %.0f MiB\n", t_mem / pow(2,20));
	printf("Memory per chunk: %.0f MiB\n", c_mem / pow(2,20));
	printf("Memory per frame: %.3f MiB\n", f_mem / pow(2,20));
	printf("Number of frames per chunk: %d\n", csz.z);


	// allocate memory on device
	CHECK(hipMalloc((float **) &dt_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &gm_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &gdev_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &nsd_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &tmp_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((float **) &norm_d, csz3 * sizeof(float)));
	CHECK(hipMalloc((uint **) &newBad_d, sizeof(uint)));

	// calculate offset for kernel
	ks2 = ksz / 2;

	// calculate striding
	S = new kso::util::stride(sz.z, csz.z, 2 * ks2, cst.z);

}

}

}

}
