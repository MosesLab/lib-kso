#include "hip/hip_runtime.h"

#include "gdev.h"


namespace kso {

namespace img {

namespace dspk {

__device__ float gdev_kern_1D(float X, float ks2, float sig){

		float x = X - ks2;

		float var = sig * sig;

		float x2 = x * x;

		return exp(-x2 / var);

}

__global__ void calc_gdev_0(float * gdev_0, float * dt, float * gm, dim3 sz, uint k_sz){

	// calculate offset for kernel
	uint ks2 = k_sz / 2;


	// retrieve sizes
	uint sz_l = sz.x;
	uint sz_y = sz.y;

	// compute stride sizes
	uint n_l = 1;
	uint n_y = n_l * sz_l;
	uint n_t = n_y * sz_y;

	// retrieve coordinates from thread and block id.
	uint l = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint t = blockIdx.z * blockDim.z + threadIdx.z;


	// initialize neighborhood mean
	float mean = 0.0;


	// convolve over spectrum
	for(uint c = 0; c < k_sz; c++){

		// calculate offset
		uint C = l - ks2 + c;

		// truncate kernel if we're over the edge
		if(C > (sz_l - 1)){
			continue;
		}

		// calculate kernel at this point
		float k_i = kfac_l * gdev_kern_1D(c, ks2, ksig_l);
//				printf("%d %f\n", c, k_i);

		// load from memory
		float gm_i = gm[n_t * t + n_y * y + n_l * C];
		float dt_i = dt[n_t * t + n_y * y + n_l * C];

		// update value of mean
		mean = mean + (k_i * gm_i * dt_i);

	}


	gdev_0[n_t * t + n_y * y + n_l * l] = mean;
}
__global__ void calc_gdev_1(float * gdev_1, float * gdev_0, dim3 sz, uint k_sz){

	// calculate offset for kernel
	uint ks2 = k_sz / 2;

	// retrieve sizes
	uint sz_l = sz.x;
	uint sz_y = sz.y;

	// compute stride sizes
	uint n_l = 1;
	uint n_y = n_l * sz_l;
	uint n_t = n_y * sz_y;

	// retrieve coordinates from thread and block id.
	uint l = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint t = blockIdx.z * blockDim.z + threadIdx.z;


	// initialize neighborhood mean
	float mean = 0.0;

	// convolve over space
	for(uint b = 0; b < k_sz; b++){

		// calculate offset
		uint B = y - ks2 + b;

		// truncate kernel if we're over the edge
		if(B > (sz_y - 1)) {
			continue;
		}

		// calculate kernel at this point
		float k_i = kfac_y * gdev_kern_1D(b, ks2, ksig_y);
//		printf("%d %f\n", b, k_i);


		// load from memory
		float gdev_i = gdev_0[n_t * t + n_y * B + n_l * l];

		// update value of mean
		mean = mean + (k_i * gdev_i);

	}


	gdev_1[n_t * t + n_y * y + n_l * l] =  mean;

}
__global__ void calc_gdev_2(float * gdev_2, float * gdev_1, float * dt, float * gm, float * norm, dim3 sz, uint k_sz){

	// calculate offset for kernel
	uint ks2 = k_sz / 2;


	// retrieve sizes
	uint sz_l = sz.x;
	uint sz_y = sz.y;
	uint sz_t = sz.z;

	// compute stride sizes
	uint n_l = 1;
	uint n_y = n_l * sz_l;
	uint n_t = n_y * sz_y;

	// retrieve coordinates from thread and block id.
	uint l = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint t = blockIdx.z * blockDim.z + threadIdx.z;


	// initialize neighborhood mean
	float mean = 0.0;

	// convolve over time
	for(uint a = 0; a < k_sz; a++){

		// calculate offsets
		uint A = t - ks2 + a;

		// truncate the kernel if we're over the edge
		if(A > (sz_t - 1)){
			continue;
		}

		// calculate kernel at this point
		float k_i = kfac_t * gdev_kern_1D(a, ks2, ksig_t);
//		printf("%f\n", k_i);


		// load from memory
		float gdev_i = gdev_1[n_t * A + n_y * y + n_l * l];

		// update value of mean
		mean = mean + (k_i * gdev_i);



	}

	float dt_i = dt[n_t * t + n_y * y + n_l * l];
	float gm_i = gm[n_t * t + n_y * y + n_l * l];
	float norm_i = norm[n_t * t + n_y * y + n_l * l];

	gdev_2[n_t * t + n_y * y + n_l * l] = mean / norm_i;
//	if(norm_i != 0.0f){
//		gdev_2[n_t * t + n_y * y + n_l * l] = mean / norm_i;
//	} else {
//		gdev_2[n_t * t + n_y * y + n_l * l] = 0.0f;
//	}

}



}

}

}
