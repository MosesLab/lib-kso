#include "hip/hip_runtime.h"

#include "util.h"

namespace kso {

namespace util {

void enum_device(){

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int device;
	for (device = 0; device < deviceCount; ++device) {
	    hipDeviceProp_t deviceProp;
	    hipGetDeviceProperties(&deviceProp, device);
	    printf("Device %d has compute capability %d.%d.\n",
	           device, deviceProp.major, deviceProp.minor);
	    printf("Device %d has %.0f MiB of memory\n", device, deviceProp.totalGlobalMem / pow(2,20));
	}

}

}

}
